/*
 * CUDA (GPU) Pathtracer based on Peter Shirley's 'Ray Tracing in One Weekend' e-book
 * series.
 *
 * Copyright (C) 2017 by Rick Weyrauch - rpweyrauch@gmail.com
 *
 * This code is licensed under the MIT license (MIT) (http://opensource.org/licenses/MIT)
 */

#include <hip/hip_math_constants.h>
#include "ptCamera.h"

Camera::Camera(float vfov, float aspect) :
    origin(0, 0, 0.),
    lowerLeftCorner(),
    horizontal(),
    vertical()
{
    float theta = vfov * HIP_PI_F / 180;
    float halfHeight = Tan(theta / 2);
    float halfWidth = aspect * halfHeight;
    lowerLeftCorner = Vector3f(-halfWidth, -halfHeight, -1);
    horizontal = Vector3f(2 * halfWidth, 0, 0);
    vertical = Vector3f(0, 2 * halfHeight, 0);
}

Camera::Camera(const Vector3f& from, const Vector3f& to, const Vector3f& vup, float vfov, float aspect, float aperture, float focal_dist, float t0, float t1) :
    origin(from),
    lowerLeftCorner(),
    horizontal(),
    vertical(),
    time0(t0),
    time1(t1)
{
    lens_radius = aperture / 2;
    float theta = vfov * HIP_PI_F / 180;
    float halfHeight = Tan(theta / 2);
    float halfWidth = aspect * halfHeight;
    w = unit_vector(from - to);
    u = unit_vector(cross(vup, w));
    v = cross(w, u);
    lowerLeftCorner = origin - halfWidth * focal_dist * u - halfHeight * focal_dist * v - focal_dist * w;
    horizontal = 2 * halfWidth * focal_dist * u;
    vertical = 2 * halfHeight * focal_dist * v;
}

bool Camera::serialize(Stream* pStream) const
{
    if (pStream == nullptr)
        return false;

    bool ok = origin.serialize(pStream);
    ok |= lowerLeftCorner.serialize(pStream);
    ok |= horizontal.serialize(pStream);
    ok |= vertical.serialize(pStream);
    ok |= u.serialize(pStream);
    ok |= v.serialize(pStream);
    ok |= w.serialize(pStream);
    ok |= pStream->write(&time0, sizeof(time0));
    ok |= pStream->write(&time1, sizeof(time1));
    ok |= pStream->write(&lens_radius, sizeof(lens_radius));

    return ok;
}

bool Camera::deserialize(Stream *pStream)
{
    if (pStream == nullptr)
        return false;

    bool ok = origin.deserialize(pStream);
    ok |= lowerLeftCorner.deserialize(pStream);
    ok |= horizontal.deserialize(pStream);
    ok |= vertical.deserialize(pStream);
    ok |= u.deserialize(pStream);
    ok |= v.deserialize(pStream);
    ok |= w.deserialize(pStream);
    ok |= pStream->read(&time0, sizeof(time0));
    ok |= pStream->read(&time1, sizeof(time1));
    ok |= pStream->read(&lens_radius, sizeof(lens_radius));

    return ok;
}

Camera* Camera::Create(Stream* pStream)
{
    if (pStream == nullptr )
        return nullptr;

    Camera* cam = new Camera;
    bool ok = cam->deserialize(pStream);
    if (!ok)
    {
        delete cam;
        cam = nullptr;
    }
    return cam;
}

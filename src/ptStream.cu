/*
 * CUDA (GPU) Pathtracer based on Peter Shirley's 'Ray Tracing in One Weekend' e-book
 * series.
 *
 * Copyright (C) 2017 by Rick Weyrauch - rpweyrauch@gmail.com
 *
 * This code is licensed under the MIT license (MIT) (http://opensource.org/licenses/MIT)
 */

#include <hip/hip_runtime.h>
#include <cstring>
#include <cstdint>
#include "ptStream.h"

Stream::Stream()
{
}

Stream::~Stream()
{
#ifndef __CUDA_ARCH__
    if (ownBuffer)
        close();
#endif
}

bool Stream::create(size_t size)
{
    if (pBuffer != nullptr)
        return false;

    hipError_t err = hipMallocManaged(&pBuffer, size);
    if (err != hipSuccess)
    {
        return false;
    }

    bufferSize = size;

    return true;
}

bool Stream::close()
{
    if (pBuffer != nullptr)
    {
        hipError_t err = hipFree(pBuffer);
        pBuffer = nullptr;
        bufferSize = 0;

        if (err != hipSuccess) return false;
    }
    return true;
}

bool Stream::write(const void* pData, size_t size)
{
    if (pBuffer == nullptr)
        return false;

    if (writeOffset + size >= bufferSize)
        return false;

    uint8_t* pDest = (uint8_t*)pBuffer + writeOffset;
    memcpy(pDest, pData, size);
    writeOffset += size;

    return true;
}

bool Stream::writeNull()
{
    int nullId = -1;
    return write(&nullId, sizeof(nullId));
}

bool Stream::read(void* pData, size_t size)
{
    if (pBuffer == nullptr)
        return false;

    if (readOffset + size >= bufferSize)
        return false;

    const uint8_t* pSrc = (uint8_t*)pBuffer + readOffset;
    memcpy(pData, pSrc, size);
    readOffset += size;

    return true;
}

#include "hip/hip_runtime.h"
/*
 * CUDA (GPU) Pathtracer based on Peter Shirley's 'Ray Tracing in One Weekend' e-book
 * series.
 *
 * Copyright (C) 2017 by Rick Weyrauch - rpweyrauch@gmail.com
 *
 * This code is licensed under the MIT license (MIT) (http://opensource.org/licenses/MIT)
 */

#include "ptCudaCommon.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <cfloat>
#include <vector>
#include "ptAABB.h"
#include "ptRectangle.h"
#include "ptRNG.h"
#include "ptSphere.h"
#include "ptHitableList.h"
#include "ptAmbientLight.h"
#include "ptRay.h"
#include "ptBVH.h"
#include "ptCamera.h"
#include "ptMaterial.h"
#include "ptMedium.h"
#include "ptProgress.h"
#include "cxxopts.hpp"

#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"

#ifdef __CUDA_ARCH__
    __device__ AmbientLight* g_ambientLight = NULL;
    __device__ Camera* g_cam;
#else
    AmbientLight* g_ambientLight = NULL;
    Camera* g_cam;
#endif


COMMON_FUNC Vector3f deNan(const Vector3f& c)
{
    Vector3f temp = c;
    if (!(temp[0] == temp[0])) temp[0] = 0;
    if (!(temp[1] == temp[1])) temp[1] = 0;
    if (!(temp[2] == temp[2])) temp[2] = 0;
    return temp;
}

/*
COMMON_FUNC Vector3f color(const Rayf& r, Hitable* world, RNG& rng, int maxDepth)
{
    Vector3f accumCol(1, 1, 1);

    Rayf currentRay(r);

    for (int depth = 0; depth < maxDepth; depth++)
    {
        HitRecord rec;
        if (world->hit(currentRay, 0.001f, FLT_MAX, rec))
        {
            Rayf scattered;
            Vector3f attenuation;
            Vector3f emitted = rec.material->emitted(rec.uv, rec.p);
            if (rec.material->scatter(currentRay, rec, attenuation, scattered, rng))
            {
                accumCol *= (emitted + attenuation);
                currentRay = scattered;
            }
            else
            {
                accumCol *= emitted;
                break;
            }
        }
        else
        {
            if (g_ambientLight)
                accumCol *= g_ambientLight->emitted(currentRay);
            else
                accumCol = Vector3f(0.0f, 0.0f, 0.0f);

            break;
        }
    }
    return accumCol;
}
*/

COMMON_FUNC Vector3f color(const Rayf& r_in, Hitable* world, Hitable* lightShape, RNG& rng, int maxDepth)
{
    Vector3f accumCol(1, 1, 1);

    Rayf currentRay(r_in);

    for (int depth = 0; depth < maxDepth; depth++)
    {
        HitRecord rec;
        if (world->hit(currentRay, 0.001f, FLT_MAX, rec, rng))
        {
            ScatterRecord srec;
            auto emitted = rec.material->emitted(currentRay, rec, rec.uv, rec.p);
            if (rec.material->scatter(currentRay, rec, srec, rng))
            {
                if (srec.isSpecular)
                {
                    accumCol *= srec.attenuation;
                    currentRay = srec.specularRay;
                }
                else
                {
                    CosinePdf pdf(rec.normal);
                    ConstPdf pdf2;
                    if (lightShape != nullptr)
                    {
                        HitablePdf plight(lightShape, rec.p);
                        MixturePdf p(&plight, &pdf);
                        auto scattered = Rayf(rec.p, p.generate(rng), currentRay.time());
                        float pdfValue = p.value(scattered.direction(), rng);
                        accumCol *= (emitted + (srec.attenuation * rec.material->scatteringPdf(currentRay, rec, scattered)) / pdfValue);
                        currentRay = scattered;
                    }
                    else
                    {
                        auto scattered = Rayf(rec.p, srec.cosinePdf ? pdf.generate(rng) : pdf2.generate(rng), currentRay.time());
                        float pdfValue = srec.cosinePdf ? pdf.value(scattered.direction(), rng) : pdf2.value(scattered.direction(), rng);
                        accumCol *= (emitted + (srec.attenuation * rec.material->scatteringPdf(currentRay, rec, scattered)) / pdfValue);
                        currentRay = scattered;
                    }
                }
            }
            else
            {
                accumCol *= emitted;
                break;
            }
        }
        else
        {
            accumCol *= g_ambientLight->emitted(currentRay);
            break;
        }
    }
    return accumCol;
}

COMMON_FUNC Vector3f render_pixel(Hitable** world, Hitable** lightShapes, int x, int y, int nx, int ny, int ns, RNG& rng, int maxDepth)
{
    Vector3f accumCol(0, 0, 0);
    for (int s = 0; s < ns; s++)
    {
        float u = (x + rng.rand()) / float(nx);
        float v = (y + rng.rand()) / float(ny);
        Rayf r = g_cam->getRay(u, v, rng);
        accumCol += deNan(color(r, *world, *lightShapes, rng, maxDepth));
    }
    accumCol /= float(ns);
    accumCol[0] = sqrtf(fmaxf(0.0f, accumCol[0]));
    accumCol[1] = sqrtf(fmaxf(0.0f, accumCol[1]));
    accumCol[2] = sqrtf(fmaxf(0.0f, accumCol[2]));

    return accumCol;
}

__global__ void render_kernel(float3* pOutImage, Hitable** world, Hitable** lightShapes, int nx, int ny, int ns, int maxDepth)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= nx || y >= ny) return;

    unsigned int i = (ny - y - 1) * nx + x; // index of current pixel (calculated using thread index)

    unsigned int seed0 = x;  // seeds for random number generator
    unsigned int seed1 = y;
    SimpleRng rng(seed0, seed1);
    Vector3f accumCol = render_pixel(world, lightShapes, x, y, nx, ny, ns, rng, maxDepth);

    pOutImage[i] = make_float3(accumCol[0], accumCol[1], accumCol[2]);
}

COMMON_FUNC void simple_spheres(Hitable** world, Hitable** lightShapes, float aspect)
{
    int i = 0;
    Hitable** list = new Hitable*[4];
    list[i++] = new Sphere(Vector3f(0.0f, 0.0f, -1.0f), 0.5f, new Lambertian(new ConstantTexture(Vector3f(0.1, 0.2, 0.5))));
    list[i++] = new Sphere(Vector3f(0.0f, -100.5f, -1.0f), 100.0f, new Lambertian(new ConstantTexture(Vector3f(0.8, 0.8, 0.0))));
    list[i++] = new Sphere(Vector3f(1, 0, -1), 0.5, new Metal(Vector3f(0.8, 0.6, 0.2), 0.3));
    list[i++] = new Sphere(Vector3f(-1, 0, -1), 0.5, new Dielectric(1.5));

    *world = new HitableList(i, list);
    *lightShapes = nullptr;

    g_cam = new Camera(Vector3f(-2, 2, 1), Vector3f(0, 0, -1), Vector3f(0, 1, 0), 90, aspect, 0.0f, 10.0f);

    delete g_ambientLight;
    g_ambientLight = new SkyAmbient();
}

COMMON_FUNC void simple_light(Hitable** world, Hitable** lightShapes, float aspect)
{
    const Vector3f lookFrom(13, 2, 3);
    const Vector3f lookAt(0, 0, 0);
    const double dist_to_focus = 10.0;
    const double aperture = 0.0;
    g_cam = new Camera(lookFrom, lookAt, Vector3f(0, 1, 0), 40, aspect, aperture, dist_to_focus);

    Texture* noise = new NoiseTexture(1.0f);
    int i = 0;
    Hitable** list = new Hitable*[4];
    list[i++] = new Sphere(Vector3f(0,-1000, 0), 1000, new Lambertian(noise));
    list[i++] = new Sphere(Vector3f(0, 2, 0), 2, new Lambertian(noise));

    list[i++] = new Sphere(Vector3f(0, 7, 0), 2, new DiffuseLight(new ConstantTexture(Vector3f(4, 4, 4))));
    list[i++] = new XYRectangle(3, 5, 1, 3, -2, new DiffuseLight(new ConstantTexture(Vector3f(4, 4, 4))));

    Hitable** lights = new Hitable*[2];
    lights[0] = new Sphere(Vector3f(0, 7, 0), 2, nullptr);
    lights[1] = new XYRectangle(3, 5, 1, 3, -2, nullptr);

    delete g_ambientLight;
    g_ambientLight = new ConstantAmbient();

    *world = new HitableList(i, list);
    *lightShapes = new HitableList(2, lights);
}

COMMON_FUNC void random_scene(Hitable** world, Hitable** lightShapes, float aspect)
{
    const Vector3f lookFrom(13, 2, 3);
    const Vector3f lookAt(0, 0, 0);
    const double dist_to_focus = 10.0;
    const double aperture = 0.0;
    g_cam = new Camera(lookFrom, lookAt, Vector3f(0, 1, 0), 20, aspect, aperture, dist_to_focus, 0.0, 1.0);

    SimpleRng rng(42, 13);

    int n = 500;
    Hitable** list = new Hitable*[n];

    int i = 0;
    Texture* checker = new CheckerTexture(new ConstantTexture(Vector3f(0.2, 0.3, 0.1)), new ConstantTexture(Vector3f(0.9, 0.9, 0.9)));
    list[i++] = new Sphere(Vector3f(0,-1000,0), 1000, new Lambertian(checker));
    for (int a = -11; a < 11; a++)
    {
        for (int b = -11; b < 11; b++)
        {
            double choose_mat = rng.rand();
            Vector3f center(a+0.9*rng.rand(),0.2,b+0.9*rng.rand());
            if ((center-Vector3f(4,0.2,0)).length() > 0.9)
            {
                if (choose_mat < 0.8) // diffuse
                {
                    list[i++] = new MovingSphere(center, center+Vector3f(0, 0.5*rng.rand(),0), 0, 1, 0.2, new Lambertian(new ConstantTexture(Vector3f(rng.rand()*rng.rand(), rng.rand()*rng.rand(), rng.rand()*rng.rand()))));
                }
                else if (choose_mat < 0.95) // metal
                {
                    list[i++] = new Sphere(center, 0.2, new Metal(Vector3f(0.5*(1+rng.rand()), 0.5*(1+rng.rand()), 0.5*rng.rand()), 0.3));
                }
                else // glass
                {
                    list[i++] = new Sphere(center, 0.2, new Dielectric(1.5));
                }
            }
        }
    }

    list[i++] = new Sphere(Vector3f(0,1,0), 1.0, new Dielectric(1.5));
    list[i++] = new Sphere(Vector3f(-4, 1, 0), 1.0, new Lambertian(new ConstantTexture(Vector3f(0.4, 0.2, 0.1))));
    list[i++] = new Sphere(Vector3f(4, 1, 0), 1.0, new Metal(Vector3f(0.7, 0.6, 0.5), 0.0));

    delete g_ambientLight;
    g_ambientLight = new SkyAmbient();

    *world = new HitableList(i, list);
    *lightShapes = nullptr;
}

COMMON_FUNC void cornell_box(Hitable **world, Hitable** lightShapes, float aspect)
{
    int i = 0;
    Hitable **list = new Hitable*[8];
    Material *red = new Lambertian( new ConstantTexture(Vector3f(0.65f, 0.05f, 0.05f)) );
    Material *white = new Lambertian( new ConstantTexture(Vector3f(0.73, 0.73, 0.73)) );
    Material *green = new Lambertian( new ConstantTexture(Vector3f(0.12, 0.45, 0.15)) );
    Material *light = new DiffuseLight( new ConstantTexture(Vector3f(15, 15, 15)) );

    list[i++] = new FlipNormals(new YZRectangle(0, 555, 0, 555, 555, green));
    list[i++] = new YZRectangle(0, 555, 0, 555, 0, red);
    list[i++] = new FlipNormals(new XZRectangle(213, 343, 227, 332, 554, light));
    list[i++] = new FlipNormals(new XZRectangle(0, 555, 0, 555, 555, white));
    list[i++] = new XZRectangle(0, 555, 0, 555, 0, white);
    list[i++] = new FlipNormals(new XYRectangle(0, 555, 0, 555, 555, white));

    //list[i++] = new Sphere(Vector3f(160, 100, 145), 100, new Dielectric(1.5));
    list[i++] = new Translate(new RotateY(new Box(Vector3f(0, 0, 0), Vector3f(165, 165, 165), white), -18), Vector3f(130, 0, 65));
    list[i++] = new Translate(new RotateY(new Box(Vector3f(0, 0, 0), Vector3f(165, 330, 165), white), 15), Vector3f(265, 0, 295));

    *world = new HitableList(i, list);

    const Vector3f lookFrom(278, 278, -800);
    const Vector3f lookAt(278, 278, 0);
    const double dist_to_focus = 10.0;
    const double aperture = 0.0;
    g_cam = new Camera(lookFrom, lookAt, Vector3f(0, 1, 0), 40, aspect, aperture, dist_to_focus);

    delete g_ambientLight;
    g_ambientLight = new SkyAmbient();

    *lightShapes = new XZRectangle(213, 343, 227, 332, 554, NULL);
}

COMMON_FUNC void cornell_box_spheres(Hitable **world, Hitable** lightShapes, float aspect)
{
    int i = 0;
    Hitable **list = new Hitable*[8];
    Material *red = new Lambertian( new ConstantTexture(Vector3f(0.65f, 0.05f, 0.05f)) );
    Material *white = new Lambertian( new ConstantTexture(Vector3f(0.73, 0.73, 0.73)) );
    Material *green = new Lambertian( new ConstantTexture(Vector3f(0.12, 0.45, 0.15)) );

    list[i++] = new Sphere(Vector3f(1e5f+1.0f, 40.8f, 81.6f), 1e5f, red);
    list[i++] = new Sphere(Vector3f(-1e5f+99.0f, 40.8f, 81.6f), 1e5f, red);

    list[i++] = new Sphere(Vector3f(50.0f, 40.8f, 1e5f), 1e5f, green);

    list[i++] = new Sphere(Vector3f(50.0f, 1e5f, 81.6f), 1e5f, white);
    list[i++] = new Sphere(Vector3f(50.0f, -1e5f + 81.6f, 81.6f), 1e5f, white);

    list[i++] = new Sphere(Vector3f(27.0f, 16.5f, 47.0f), 16.5f, white);
    list[i++] = new Sphere(Vector3f(73.0f, 16.5f, 78.0f), 16.5f, white);

    *world = new HitableList(i, list);

    const Vector3f lookFrom(278, 278, -800);
    const Vector3f lookAt(278, 278, 0);
    const double dist_to_focus = 10.0;
    const double aperture = 0.0;
    g_cam = new Camera(lookFrom, lookAt, Vector3f(0, 1, 0), 40, aspect, aperture, dist_to_focus);

    delete g_ambientLight;
    g_ambientLight = new ConstantAmbient();

}

COMMON_FUNC void final(Hitable **world, Hitable** lightShapes, float aspect)
{
    const Vector3f lookFrom(478, 278, -600); //(278, 278, -800); //(13, 2, 3);
    const Vector3f lookAt(278, 278, 0); //(0, 1, 0);
    const float dist_to_focus = 10.0f;
    const float aperture = 0.0f;
    g_cam = new Camera(lookFrom, lookAt, Vector3f(0, 1, 0), 40, aspect, aperture, dist_to_focus);

    int nb = 20;

    Material* white = new Lambertian(new ConstantTexture(Vector3f(0.73, 0.73, 0.73)));
    Material* ground = new Lambertian(new ConstantTexture(Vector3f(0.48, 0.83, 0.53)));
    Hitable** boxList = new Hitable*[nb*nb];

    SimpleRng rng(42, 13);

    int bi = 0;
    for (int i = 0; i < nb; i++)
    {
        for (int j = 0; j < nb; j++)
        {
            float w = 100;
            float x0 = -1000 + i*w;
            float z0 = -1000 + j*w;
            float y0 = 0;
            float x1 = x0 + w;
            float y1 = 100*(rng.rand()+0.01f);
            float z1 = z0 + w;
            boxList[bi++] = new Box(Vector3f(x0, y0, z0), Vector3f(x1, y1, z1), ground);
        }
    }

    int i = 0;
    Hitable **list = new Hitable*[12];
    list[i++] = new BVH(boxList, bi, 0, 1, rng);
    Material* light = new DiffuseLight(new ConstantTexture(Vector3f(6, 6, 6)));
    list[i++] = new FlipNormals(new XZRectangle(123, 423, 147, 412, 554, light));
    Vector3f center(400, 400, 200);
    list[i++] = new MovingSphere(center, center+Vector3f(30, 0, 0), 0, 1, 50, new Lambertian(new ConstantTexture(Vector3f(0.7, 0.3, 0.1))));
    list[i++] = new Sphere(Vector3f(260, 150, 45), 50, new Dielectric(1.5));
    list[i++] = new Sphere(Vector3f(0, 150, 145), 50, new Metal(Vector3f(0.8, 0.8, 0.9), 10));
    Hitable* boundary = new Sphere(Vector3f(360, 150, 145), 70, new Dielectric(1.5));
    list[i++] = boundary;
    list[i++] = new ConstantMedium(boundary, 0.02, new ConstantTexture(Vector3f(0.2, 0.4, 0.9)));
    boundary = new Sphere(Vector3f(0, 0, 0), 5000, new Dielectric(1.5));
    list[i++] = new ConstantMedium(boundary, 0.0001, new ConstantTexture(Vector3f(1.0, 1.0, 1.0)));
    //int nx, ny, nz;
    //unsigned char* tex_data = stbi_load("earthmap.jpg", &nx, &ny, &nz, 0);
    //Material* emat = new Lambertian(new ImageTexture(tex_data, nx, ny));
    //list[i++] = new Sphere(Vector3f(400, 200, 400), 100, emat);
    Texture* pertext = new NoiseTexture(0.1);
    list[i++] = new Sphere(Vector3f(220, 280, 300), 80, new Lambertian(pertext));
    int ns = 1000;
    Hitable** boxList2 = new Hitable*[ns];

    for (int j = 0; j < ns; j++)
    {
        boxList2[j] = new Sphere(Vector3f(165*rng.rand(), 165*rng.rand(), 165*rng.rand()), 10, white);
    }
    list[i++] = new Translate(new RotateY(new BVH(boxList2, ns, 0.0f, 1.0f, rng), 15), Vector3f(-100, 270, 395));

    *lightShapes = new XZRectangle(123, 423, 147, 412, 554, nullptr);
    //lights.push_back(new Sphere(Vector3(360, 150, 145), 70, nullptr));
    //lights.push_back(new Sphere(Vector3(0, 0, 0), 5000, nullptr));

    *world = new HitableList(i, list);
}

__global__ void allocate_world_kernel(Hitable** world, Hitable** lightShapes, float aspect)
{
    cornell_box(world, lightShapes, aspect);
}

void writeImage(const std::string& outFile, const Vector3f* outImage, int nx, int ny)
{
    auto extStart = outFile.rfind('.');
    if (extStart != std::string::npos)
    {
        extStart++;
        std::string ext = outFile.substr(extStart);
        if (ext == "ppm")
        {
            std::ofstream of(outFile.c_str());
            if (of.is_open())
            {
                of << "P3\n" << nx << " " << ny << "\n255\n";

                for (int i = 0; i < nx * ny; i++)
                {
                    Vector3f col = outImage[i];

                    int ir = int(255.99 * col[0]);
                    int ig = int(255.99 * col[1]);
                    int ib = int(255.99 * col[2]);

                    of << ir << " " << ig << " " << ib << "\n";
                }
            }
            of.close();
        }
        else if (ext == "hdr")
        {
            stbi_write_hdr(outFile.c_str(), nx, ny, 3, (const float*)outImage);
        }
        else
        {
            unsigned char* outBytes = new unsigned char[nx * ny * 3];
            unsigned char* currentOut = outBytes;
            for (int i = 0; i < nx * ny; i++)
            {
                const Vector3f& col = outImage[i];
                int ir = Clamp(int(255.99 * col[0]), 0, 255);
                int ig = Clamp(int(255.99 * col[1]), 0, 255);
                int ib = Clamp(int(255.99 * col[2]), 0, 255);
                *currentOut++ = (unsigned char)ir;
                *currentOut++ = (unsigned char)ig;
                *currentOut++ = (unsigned char)ib;
            }
            if (ext == "png")
                stbi_write_png(outFile.c_str(), nx, ny, 3, outBytes, nx * 3);
            else if (ext == "tga")
                stbi_write_tga(outFile.c_str(), nx, ny, 3, outBytes);
            else if (ext == "bmp")
                stbi_write_bmp(outFile.c_str(), nx, ny, 3, outBytes);

            delete[] outBytes;
        }
    }
}

void renderLine(int line, Vector3f* outLine, int nx, int ny, int ns, Camera& cam, Hitable* world, Hitable* lightShapes, RNG& rng, int maxDepth)
{
    for (int x = 0; x < nx; x++)
    {
        outLine[x] = render_pixel(&world, &lightShapes, x, line, nx, ny, ns, rng, maxDepth);
    }
}

int main(int argc, char** argv)
{
    cxxopts::Options options("pathtracer", "Implementation of Peter Shirley's Raytracing in One Weekend book series.");
    options.add_options()
        ("q,quick", "Quick render.")
        ("c,cpu", "Render on CPU.")
        ("w,width", "Output width.", cxxopts::value<int>())
        ("h,height", "Output height.", cxxopts::value<int>())
        ("n,numsamples", "Number of sample rays per pixel.", cxxopts::value<int>())
        ("t,threads", "Number of render threads.", cxxopts::value<int>())
        ("d,maxdepth", "Maximum ray bounces.", cxxopts::value<int>())
        ("f,file", "Output filename.", cxxopts::value<std::string>());

    options.parse(argc, argv);

    bool quick = options.count("quick") > 0;
    int ns = 100;
    int nx = 128 * 4;
    int ny = 128 * 4;
    bool cpu = options.count("cpu") > 0;
    int numThreads = 1;
    int maxDepth = 25;

    std::string outFile("outputImage.ppm");

    if (options.count("width"))
        nx = options["width"].as<int>();
    if (options.count("height"))
        ny = options["height"].as<int>();
    if (options.count("numsamples"))
        ns = options["numsamples"].as<int>();
    if (options.count("maxdepth"))
        maxDepth = options["maxdepth"].as<int>();
    if (options.count("file"))
        outFile = options["file"].as<std::string>();
    if (options.count("threads"))
        numThreads = options["numthreads"].as<int>();

    if (quick)
    {
        nx /= 8;
        ny /= 8;
        ns /= 16;
    }

    const float aspect = float(nx)/float(ny);

    Vector3f* outImage = new Vector3f[nx * ny];

    if (!cpu)
    {
        float3* pOutImage = NULL;
        hipMalloc(&pOutImage, nx * ny * sizeof(float3));

        Hitable** world = NULL;
        hipMalloc(&world, sizeof(Hitable**));

        Hitable** lightShapes = NULL;
        hipMalloc(&lightShapes, sizeof(Hitable**));

        std::cerr << "Allocating world...";
        allocate_world_kernel<<<1, 1>>>(world, lightShapes, aspect);
        hipError_t err = hipDeviceSynchronize();
        std::cerr << "done" << std::endl;
        if (err != hipSuccess)
        {
            std::cerr << "Failed to allocate world in GPU memory.  Error: " << hipGetErrorName(err) << " Desc: " << hipGetErrorString(err) << std::endl;
            return EXIT_FAILURE;
        }

        dim3 block(8, 8, 1);
        dim3 grid(IDIVUP(nx, block.x), IDIVUP(ny, block.y), 1);
        std::cerr << "Rendering world...";
        render_kernel<<<grid, block>>>(pOutImage, world, lightShapes, nx, ny, ns, maxDepth);
        err = hipDeviceSynchronize();
        std::cerr << "done" << std::endl;
        if (err != hipSuccess)
        {
            std::cerr << "Failed to render on GPU.  Error: " << hipGetErrorName(err) << " Desc: " << hipGetErrorString(err) << std::endl;
            return EXIT_FAILURE;
        }

        hipMemcpy(outImage, pOutImage, nx*ny*sizeof(Vector3f), hipMemcpyDeviceToHost);
        hipFree(pOutImage);
        hipFree(lightShapes);
        hipFree(world);
    }
    else
    {
        Camera cam;
        Hitable* world = NULL;
        Hitable* lightShapes = NULL;
        cornell_box(&world, &lightShapes, aspect);// cornellBox(); // simpleLight(); //randomScene(); //

        Stream* pStream = new Stream();
        pStream->create(1024 * 1024 * 16);

        Hitable* pSphere = new Sphere(Vector3f(0, 1, 2), 3, new Lambertian(nullptr));
        bool ok = pSphere->serialize(pStream);
        if (ok)
        {
            Hitable* clone = Hitable::Create(pStream);
        }
        pStream->close();
        delete pStream;

        unsigned int seed0 = 42;
        unsigned int seed1 = 13;
        DRandRng rng(seed0);//, seed1);

        Progress progress(nx*ny, "PathTracers");

        #pragma omp parallel for if(numThreads)
        for (int j = 0; j < ny; j++)
        {
            Vector3f* outLine = outImage + (nx * j);
            const int line = ny - j - 1;
            renderLine(line, outLine, nx, ny, ns, cam, world, lightShapes, rng, maxDepth);

            #pragma omp critical(progress)
            {
                progress.update(nx);
            }
        }

        progress.completed();
    }

    writeImage(outFile, outImage, nx, ny);
    delete[] outImage;
    std::cerr << "Done." << std::endl;

    return EXIT_SUCCESS;
}

#include "hip/hip_runtime.h"
/*
 * CUDA (GPU) Pathtracer based on Peter Shirley's 'Ray Tracing in One Weekend' e-book
 * series.
 *
 * Copyright (C) 2017 by Rick Weyrauch - rpweyrauch@gmail.com
 *
 * This code is licensed under the MIT license (MIT) (http://opensource.org/licenses/MIT)
 */

#include "ptCudaCommon.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <cfloat>
#include <vector>
#include "ptAABB.h"
#include "ptRectangle.h"
#include "ptSphere.h"
#include "ptHitableList.h"
#include "ptAmbientLight.h"
#include "ptRay.h"
#include "ptCamera.h"
#include "ptMaterial.h"
#include "ptProgress.h"
#include "cxxopts.hpp"

#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

const int MAX_DEPTH = 25;

#ifdef __CUDA_ARCH__
    __device__ AmbientLight* g_ambientLight = NULL;
    __device__ Camera g_cam;
#else
    AmbientLight* g_ambientLight = NULL;
    Camera g_cam;
#endif

COMMON_FUNC Vector3f color_nr(const Ray<float>& r, Hitable* world, Hitable* lightShape, RNG* rng)
{
    Vector3f accumCol(1, 1, 1);

    Ray<float> currentRay(r);

    for (int depth = 0; depth < MAX_DEPTH; depth++)
    {
        HitRecord rec;
        if (world->hit(r, 0.001f, FLT_MAX, rec))
        {
            ScatterRecord srec;
            auto emitted = rec.material->emitted(r, rec, rec.uv, rec.p);
            if (rec.material->scatter(r, rec, srec, rng))
            {
                if (srec.isSpecular)
                {
                    accumCol *= srec.attenuation;
                    currentRay = srec.specularRay;
                }
                else
                {
                    if (lightShape != nullptr)
                    {
                        HitablePdf plight(lightShape, rec.p);
                        MixturePdf p(&plight, srec.pdf);
                        auto scattered = Ray<float>(rec.p, p.generate(rng), r.time());
                        float pdfValue = p.value(scattered.direction());
                        delete srec.pdf;
                        accumCol *= (emitted + (srec.attenuation * rec.material->scatteringPdf(currentRay, rec, scattered)) / pdfValue);
                        currentRay = scattered;
                    }
                    else
                    {
                        auto scattered = Ray<float>(rec.p, srec.pdf->generate(rng), r.time());
                        float pdfValue = srec.pdf->value(scattered.direction());
                        delete srec.pdf;
                        accumCol *= (emitted + (srec.attenuation * rec.material->scatteringPdf(currentRay, rec, scattered)) / pdfValue);
                        currentRay = scattered;
                    }
                }
            }
            else
            {
                accumCol *= emitted;
                break;
            }
        }
        else
        {
            accumCol *= g_ambientLight->emitted(r);
            break;
        }
    }
    return accumCol;
}

__global__ void render_kernel(float3* pOutImage, Hitable** world, Hitable** lightShape, int nx, int ny, int ns)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= nx || y >= ny) return;

    unsigned int i = (ny - y - 1) * nx + x; // index of current pixel (calculated using thread index)

    unsigned int seed0 = x;  // seeds for random number generator
    unsigned int seed1 = y;
    SimpleRng rng(seed0, seed1);

    Vector3<float> accumCol(0, 0, 0);
    for (int s = 0; s < ns; s++)
    {
        float u = (x + rng.rand()) / float(nx);
        float v = (y + rng.rand()) / float(ny);
        Ray<float> r = g_cam.getRay(u, v, &rng);
        accumCol += color_nr(r, *world, *lightShape, &rng);
    }
    accumCol /= float(ns);

    pOutImage[i] = make_float3(sqrtf(accumCol[0]), sqrtf(accumCol[1]), sqrtf(accumCol[2]));
}

__global__ void allocate_world_kernel(Hitable** world, Hitable** lightShape, float aspect)
{
    int i = 0;
    Hitable** list = new Hitable*[4];
    list[i++] = new Sphere(Vector3f(0.0f, 0.0f, -1.0f), 0.5f, new Lambertian(new ConstantTexture(Vector3f(0.1, 0.2, 0.5))));
    list[i++] = new Sphere(Vector3f(0.0f, -100.5f, -1.0f), 100.0f, new Lambertian(new ConstantTexture(Vector3f(0.8, 0.8, 0.0))));
    list[i++] = new Sphere(Vector3f(1, 0, -1), 0.5, new Metal(Vector3f(0.8, 0.6, 0.2), 0.3));
    list[i++] = new Sphere(Vector3f(-1, 0, -1), 0.5, new Dielectric(1.5));

    *world = new HitableList(i, list);

    g_cam = Camera(Vector3f(-2, 2, 1), Vector3f(0, 0, -1), Vector3f(0, 1, 0), 90, aspect, 0.0f, 10.0f);

    g_ambientLight = new SkyAmbient();

    *lightShape = NULL;
}

__global__ void cornell_box_kernel(Hitable **world, Hitable** lightShape, float aspect)
{
    int i = 0;
    Hitable **list = new Hitable*[8];
    Material *red = new Lambertian( new ConstantTexture(Vector3f(0.65f, 0.05f, 0.05f)) );
    Material *white = new Lambertian( new ConstantTexture(Vector3f(0.73, 0.73, 0.73)) );
    Material *green = new Lambertian( new ConstantTexture(Vector3f(0.12, 0.45, 0.15)) );
    Material *light = new DiffuseLight( new ConstantTexture(Vector3f(15, 15, 15)) );
    Material* aluminum = new Metal(Vector3f(0.8, 0.85, 0.88), 0.0);

    list[i++] = new FlipNormals(new YZRectangle(0, 555, 0, 555, 555, green));
    list[i++] = new YZRectangle(0, 555, 0, 555, 0, red);
    list[i++] = new FlipNormals(new XZRectangle(213, 343, 227, 332, 554, light));
    list[i++] = new FlipNormals(new XZRectangle(0, 555, 0, 555, 555, white));
    list[i++] = new XZRectangle(0, 555, 0, 555, 0, white);
    list[i++] = new FlipNormals(new XYRectangle(0, 555, 0, 555, 555, white));

    list[i++] = new Translate(new RotateY(new Box(Vector3f(0, 0, 0), Vector3f(165, 165, 165), white), -18), Vector3f(130, 0, 65));
    list[i++] = new Translate(new RotateY(new Box(Vector3f(0, 0, 0), Vector3f(165, 330, 165), aluminum), 15), Vector3f(265, 0, 295));

    *world = new HitableList(i, list);

    const Vector3f lookFrom(278, 278, -800);
    const Vector3f lookAt(278, 278, 0);
    const double dist_to_focus = 10.0;
    const double aperture = 0.0;
    g_cam = Camera(lookFrom, lookAt, Vector3f(0, 1, 0), 40, aspect, aperture, dist_to_focus);

    g_ambientLight = new ConstantAmbient();

    *lightShape = new XZRectangle(213, 343, 227, 332, 554, NULL);
}

void writeImage(const std::string& outFile, const Vector3f* outImage, int nx, int ny)
{
    auto extStart = outFile.rfind('.');
    if (extStart != std::string::npos)
    {
        extStart++;
        std::string ext = outFile.substr(extStart);
        if (ext == "ppm")
        {
            std::ofstream of(outFile.c_str());
            if (of.is_open())
            {
                of << "P3\n" << nx << " " << ny << "\n255\n";

                for (int i = 0; i < nx * ny; i++)
                {
                    Vector3f col = outImage[i];

                    int ir = int(255.99 * col[0]);
                    int ig = int(255.99 * col[1]);
                    int ib = int(255.99 * col[2]);

                    of << ir << " " << ig << " " << ib << "\n";
                }
            }
            of.close();
        }
        else if (ext == "hdr")
        {
            stbi_write_hdr(outFile.c_str(), nx, ny, 3, (const float*)outImage);
        }
        else
        {
            unsigned char* outBytes = new unsigned char[nx * ny * 3];
            unsigned char* currentOut = outBytes;
            for (int i = 0; i < nx * ny; i++)
            {
                const Vector3f& col = outImage[i];
                int ir = Clamp(int(255.99 * col[0]), 0, 255);
                int ig = Clamp(int(255.99 * col[1]), 0, 255);
                int ib = Clamp(int(255.99 * col[2]), 0, 255);
                *currentOut++ = (unsigned char)ir;
                *currentOut++ = (unsigned char)ig;
                *currentOut++ = (unsigned char)ib;
            }
            if (ext == "png")
                stbi_write_png(outFile.c_str(), nx, ny, 3, outBytes, nx * 3);
            else if (ext == "tga")
                stbi_write_tga(outFile.c_str(), nx, ny, 3, outBytes);
            else if (ext == "bmp")
                stbi_write_bmp(outFile.c_str(), nx, ny, 3, outBytes);

            delete[] outBytes;
        }
    }
}

inline Vector3f deNan(const Vector3f& c)
{
    Vector3f temp = c;
    if (!(temp[0] == temp[0])) temp[0] = 0;
    if (!(temp[1] == temp[1])) temp[1] = 0;
    if (!(temp[2] == temp[2])) temp[2] = 0;
    return temp;
}

void renderLine(int line, Vector3f* outLine, int nx, int ny, int ns, Camera& cam, Hitable* world, Hitable* lightShapes, RNG* rng)
{
    for (int x = 0; x < nx; x++)
    {
        Vector3f col(0, 0, 0);
        for (int s = 0; s<ns; s++)
        {
            auto u = (x + rng->rand()) / float(nx);
            auto v = (line + rng->rand()) / float(ny);
            Rayf r = cam.getRay(u, v, rng);
            col += deNan(color_nr(r, world, lightShapes, rng));
        }
        col /= double(ns);
        outLine[x] = Vector3f(sqrt(std::max(0.0f, col[0])), sqrt(std::max(0.0f, col[1])), sqrt(std::max(0.0f, col[2])));
    }
}

Hitable* cornellBox(float aspect, Camera& camera, std::vector<Hitable*>& lights)
{
    const Vector3f lookFrom(278, 278, -800);
    const Vector3f lookAt(278, 278, 0);
    const float dist_to_focus = 10.0;
    const float aperture = 0.0;
    camera = Camera(lookFrom, lookAt, Vector3f(0, 1, 0), 40, aspect, aperture, dist_to_focus);

    std::vector<Hitable*> list;

    Material* red = new Lambertian(new ConstantTexture(Vector3f(0.65, 0.05, 0.05)));
    Material* white = new Lambertian(new ConstantTexture(Vector3f(0.73, 0.73, 0.73)));
    Material* green = new Lambertian(new ConstantTexture(Vector3f(0.12, 0.45, 0.15)));
    Material* light = new DiffuseLight(new ConstantTexture(Vector3f(15, 15, 15)));
    Material* aluminum = new Metal(Vector3f(0.8, 0.85, 0.88), 0.0);
    Material* glass = new Dielectric(1.5);

    list.push_back(new FlipNormals(new YZRectangle(0, 555, 0, 555, 555, green)));
    list.push_back(new YZRectangle(0, 555, 0, 555, 0, red));
    list.push_back(new FlipNormals(new XZRectangle(213, 343, 227, 332, 554, light)));
    list.push_back(new FlipNormals(new XZRectangle(0, 555, 0, 555, 555, white)));
    list.push_back(new XZRectangle(0, 555, 0, 555, 0, white));
    list.push_back(new FlipNormals(new XYRectangle(0, 555, 0, 555, 555, white)));

    list.push_back(new Translate(new RotateY(new Box(Vector3f(0, 0, 0), Vector3f(165, 165, 165), white), -18), Vector3f(130, 0, 65)));
    list.push_back(new Translate(new RotateY(new Box(Vector3f(0, 0, 0), Vector3f(165, 330, 165), aluminum), 15), Vector3f(265, 0, 295)));
    //list.push_back(new Translate(new Box(Vector3(0, 0, 0), Vector3(165, 330, 165), aluminum), Vector3(265, 0, 295)));
    //list.push_back(new Sphere(Vector3(190, 90, 190), 90, glass));

    //list.push_back(new Translate(new Box(Vector3(0, 0, 0), Vector3(165, 165, 165), white), Vector3(130, 0, 65)));
    //list.push_back(new Translate(new Box(Vector3(0, 0, 0), Vector3(165, 330, 165), white), Vector3(265, 0, 295)));

    //Hitable* b1 = new Translate(new RotateY(new Box(Vector3(0, 0, 0), Vector3(165, 165, 165), white), -18), Vector3(130, 0, 65));
    //Hitable* b2 = new Translate(new RotateY(new Box(Vector3(0, 0, 0), Vector3(165, 330, 165), white), 15), Vector3(265, 0, 295));

    //list.push_back(new ConstantMedium(b1, 0.01, new ConstantTexture(Vector3(1, 1, 1))));
    //list.push_back(new ConstantMedium(b2, 0.01, new ConstantTexture(Vector3(0, 0, 0))));

    lights.push_back(new XZRectangle(213, 343, 227, 332, 554, nullptr));

    g_ambientLight = new SkyAmbient();

    return new HitableList(list.size(), list.data());
}

int main(int argc, char** argv)
{
    cxxopts::Options options("pathtracer", "Implementation of Peter Shirley's Raytracing in One Weekend book series.");
    options.add_options()
        ("q,quick", "Quick render.")
        ("c,cpu", "Render on CPU.")
        ("w,width", "Output width.", cxxopts::value<int>())
        ("h,height", "Output height.", cxxopts::value<int>())
        ("n,numsamples", "Number of sample rays per pixel.", cxxopts::value<int>())
        ("t,threads", "Number of render threads.", cxxopts::value<int>())
        ("f,file", "Output filename.", cxxopts::value<std::string>());

    options.parse(argc, argv);

    bool quick = options.count("quick") > 0;
    int ns = 100;
    int nx = 128 * 4;
    int ny = 128 * 4;
    bool cpu = options.count("cpu") > 0;
    int numThreads = 1;

    std::string outFile("outputImage.ppm");

    if (options.count("width"))
        nx = options["width"].as<int>();
    if (options.count("height"))
        ny = options["height"].as<int>();
    if (options.count("numsamples"))
        ns = options["numsamples"].as<int>();
    if (options.count("file"))
        outFile = options["file"].as<std::string>();
    if (options.count("threads"))
        numThreads = options["numthreads"].as<int>();

    if (quick)
    {
        nx /= 8;
        ny /= 8;
        ns /= 16;
    }

    const float aspect = float(nx)/float(ny);

    if (!cpu)
    {
        float3* pOutImage = NULL;
        hipMalloc(&pOutImage, nx * ny * sizeof(float3));

        Hitable** world = NULL;
        hipMalloc(&world, sizeof(Hitable**));

        Hitable** lightShape = NULL;
        hipMalloc(&lightShape, sizeof(Hitable**));

        std::cerr << "Allocating world...";
        allocate_world_kernel<<<1, 1>>>(world, lightShape, aspect);
        hipError_t err = hipDeviceSynchronize();
        std::cerr << "done" << std::endl;
        if (err != hipSuccess)
        {
            std::cerr << "Failed to allocate world in GPU memory.  Error: " << hipGetErrorName(err) << " Desc: " << hipGetErrorString(err) << std::endl;
            return EXIT_FAILURE;
        }

        dim3 block(8, 8, 1);
        dim3 grid(IDIVUP(nx, block.x), IDIVUP(ny, block.y), 1);
        std::cerr << "Rendering world...";
        render_kernel<<<grid, block>>>(pOutImage, world, lightShape, nx, ny, ns);
        err = hipDeviceSynchronize();
        std::cerr << "done" << std::endl;
        if (err != hipSuccess)
        {
            std::cerr << "Failed to render on GPU.  Error: " << hipGetErrorName(err) << " Desc: " << hipGetErrorString(err) << std::endl;
            return EXIT_FAILURE;
        }

        Vector3f* pTemp = new Vector3f[nx * ny];
        hipMemcpy(pTemp, pOutImage, nx*ny*sizeof(float3), hipMemcpyDeviceToHost);
        hipFree(pOutImage);

        writeImage(outFile, pTemp, nx, ny);
        delete[] pTemp;
    }
    else
    {
        Camera cam;
        const double aspect = double(nx)/double(ny);
        std::vector<Hitable*> lights;
        Hitable* world = cornellBox(aspect, cam, lights);// cornellBox(); // simpleLight(); //randomScene(); //
        HitableList* lightShapes = nullptr;
        if (!lights.empty())
            lightShapes = new HitableList(lights.size(), lights.data());
        //Hitable* lightShape = new XZRectangle(213, 343, 227, 332, 554, nullptr);
        //Hitable* glassSphere = new Sphere(Vector3(190, 90, 190), 90, nullptr);
        //lights.push_back(lightShape);
        //lights.push_back(glassSphere);
        //HitableList* lightShapes = new HitableList(lights);

        Vector3f* outImage = new Vector3f[nx * ny];

        Progress progress(nx*ny, "PathTracers");

        DRandRng* rng = new DRandRng(42);

        int index = 0;
        #pragma omp parallel for if(numThreads)
        for (int j = 0; j < ny; j++)
        {
            Vector3f* outLine = outImage + (nx * j);
            const int line = ny - j - 1;
            renderLine(line, outLine, nx, ny, ns, cam, world, lightShapes, rng);

            #pragma omp critical(progress)
            {
                progress.update(nx);
            }
        }

        progress.completed();

        writeImage(outFile, outImage, nx, ny);

        delete rng;
        delete[] outImage;
    }

    std::cerr << "Done." << std::endl;

    return EXIT_SUCCESS;
}

#include "hip/hip_runtime.h"
/*
 * CUDA (GPU) Pathtracer based on Peter Shirley's 'Ray Tracing in One Weekend' e-book
 * series.
 *
 * Copyright (C) 2017 by Rick Weyrauch - rpweyrauch@gmail.com
 *
 * This code is licensed under the MIT license (MIT) (http://opensource.org/licenses/MIT)
 */

#include "ptTriangle.h"
#include "ptMaterial.h"

Triangle::Triangle(const Vector3f& v0, const Vector2f& t0,
         const Vector3f& v1, const Vector2f& t1,
         const Vector3f& v2, const Vector2f& t2,
         Material* mtl) :
    v0(v0),
    v1(v1),
    v2(v2),
    t0(t0),
    t1(t1),
    t2(t2),
    material(mtl)
    {
        calcBounds();
    }

bool Triangle::hit(const Rayf& r, float t_min, float t_max, HitRecord& rec, RNG& rng) const
{
    //
    // Tomas Moller and Ben Trumbore, "Fast Minimum Storage Ray-Triangle Intersection,"
    // Journal of Graphics Tools, Vol. 2, No. 1, pp. 21-28, 1997.
    //

    // Find vectors for two edges sharing v0.
    Vector3f edge1(v1 - v0);
    Vector3f edge2(v2 - v0);

    // Begin calculating determinant - also used to calculate U parameter.
    Vector3f pvec = cross(r.direction(), edge2);

    // If determinant is near zero, ray lies in plane of triangle.
    auto det = dot(edge1, pvec);

    if (det < 0.0001f)
        return false;

    // Calculate distance from v0 to ray origin.
    Vector3f tvec(r.origin() - v0);

    // calculate U parameter and test bounds.
    float u = dot(tvec, pvec);
    if (u < 0 || u > det)
        return false;

    // Prepare to test V parameter.
    Vector3f qvec = cross(tvec, edge1);

    // Calculate V parameter and test bounds.
    auto v = dot(r.direction(), qvec);
    if (v < 0 || u + v > det)
        return false;

    // Calculate t, scale parameters, ray intersects triangle.
    float t = dot(edge2, qvec);
    if (t < t_min || t > t_max) return false;

    const auto inv_det = 1 / det;
    u *= inv_det;
    v *= inv_det;

    rec.t = t;
    rec.p = (1 - u - v) * v0 + u * v1 + v * v2;
    rec.normal = cross(edge1, edge2);
    rec.normal.make_unit_vector();
    rec.material = material;

    Vector3f bary(1.0 - u - v, u, v);
    calcTexCoord(bary, rec.uv);

    return true;
}

bool Triangle::bounds(float t0, float t1, AABB<float>& bbox) const
{
    bbox = this->bbox;
    return true;
}

float Triangle::area() const
{
    Vector3f u(v1 - v0);
    Vector3f v(v2 - v0);
    Vector3f uv = cross(u, v);
    return 0.5f * uv.length();
}

void Triangle::calcTexCoord(const Vector3f& bary, Vector2f& uv) const
{
    uv = t0 * bary.x() + t1 * bary.y() + t2 * bary.z();
}

void Triangle::calcBounds()
{
    Vector3f bmin{}, bmax{};
    for (int i = 0; i < 3; i++)
    {
        bmin[i] = Min(v0[i]-0.0001f, Min(v1[i]-0.0001f, v2[i]-0.0001f));
        bmax[i] = Max(v0[i]+0.0001f, Max(v1[i]+0.0001f, v2[i]+0.0001f));
    }
    bbox = AABB<float>(bmin, bmax);
}

bool Triangle::serialize(Stream *pStream) const
{
    if (pStream == nullptr)
        return false;

    const int id = typeId();
    bool ok = pStream->write(&id, sizeof(id));
    ok |= v0.serialize(pStream);
    ok |= v1.serialize(pStream);
    ok |= v2.serialize(pStream);
    ok |= t0.serialize(pStream);
    ok |= t1.serialize(pStream);
    ok |= t2.serialize(pStream);
    ok |= material->serialize(pStream);
    ok |= bbox.serialize(pStream);

    return ok;
}

bool Triangle::deserialize(Stream *pStream)
{
    if (pStream == nullptr)
        return false;

    bool ok = v0.deserialize(pStream);
    ok |= v1.deserialize(pStream);
    ok |= v2.deserialize(pStream);
    ok |= t0.deserialize(pStream);
    ok |= t1.deserialize(pStream);
    ok |= t2.deserialize(pStream);
    material = Material::Create(pStream);
    ok |= bbox.deserialize(pStream);

    return ok;
}


bool TriangleMesh::hit(const Rayf& r, float t_min, float t_max, HitRecord& rec, RNG& rng) const
{
    return false;
}

bool TriangleMesh::hit(const Rayf& ray, const TriangleFast& accel, float& tHit, Vector3f& bary) const
{
    //
    // "Real Time Ray Tracing and Interactive Global Illumination", Ingo Wald:
    // http://www.mpi-sb.mpg.de/~wald/PhD/
    //
    // Jakko Bikker
    // http://www.flipcode.com/articles/article_raytrace07.shtml
    //
    static int axisModulo[] = { 0, 1, 2, 0, 1 };
    const int ku = axisModulo[accel.m_k+1];
    const int kv = axisModulo[accel.m_k+2];

    const float nd = 1 / (ray.direction()[accel.m_k] + accel.m_nu * ray.direction()[ku] + accel.m_nv * ray.direction()[kv]);
    float t = (accel.m_nd - ray.origin()[accel.m_k] - accel.m_nu * ray.origin()[ku] - accel.m_nv * ray.origin()[kv]) * nd;

    if (t < 0)
    {
        return false;
    }

    const float hu = ray.origin()[ku] + t * ray.direction()[ku] - accel.m_v0[ku];
    const float hv = ray.origin()[kv] + t * ray.direction()[kv] - accel.m_v0[kv];

    const float u = hv * accel.m_bnu + hu * accel.m_bnv;
    if (u < 0)
        return false;

    const float v = hu * accel.m_cnu + hv * accel.m_cnv;
    if (v < 0)
        return false;

    if (u + v > 1)
        return false;

    tHit = t;
    bary = Vector3f(1 - u - v, u, v);

    return true;
}

bool TriangleMesh::bounds(float t0, float t1, AABB<float>& bbox) const
{
    return false;
}

void TriangleMesh::addVertex(const Vector3f& p, const Vector3f& n, const Vector2f& tex)
{
    verts.push_back(p);
    normals.push_back(n);
    texCoords.push_back(tex);
}

void TriangleMesh::complete()
{
    delete[] triAccel;
    count = triangles.size();
    triAccel = new TriangleFast[count];

    int i = 0;
    for (const auto ip : triangles)
    {
        TriangleFast triFast(verts[ip.i0], verts[ip.i1],  verts[ip.i2]);
        triAccel[i++] = triFast;
    }
}

bool TriangleMesh::serialize(Stream *pStream) const
{
    if (pStream == nullptr)
        return false;

    const int id = typeId();
    bool ok = pStream->write(&id, sizeof(id));
    ok |= pStream->write(&count, sizeof(count));
    for (int i = 0; i < count && ok; i++)
    {
        ok |= triAccel[i].m_v0.serialize(pStream);
        ok |= pStream->write(&triAccel[i].m_nu, sizeof(float));
        ok |= pStream->write(&triAccel[i].m_nv, sizeof(float));
        ok |= pStream->write(&triAccel[i].m_nd, sizeof(float));
        ok |= pStream->write(&triAccel[i].m_k, sizeof(int));
        ok |= pStream->write(&triAccel[i].m_bnu, sizeof(float));
        ok |= pStream->write(&triAccel[i].m_bnv, sizeof(float));
        ok |= pStream->write(&triAccel[i].m_cnu, sizeof(float));
        ok |= pStream->write(&triAccel[i].m_cnv, sizeof(float));
    }

    return ok;
}

bool TriangleMesh::deserialize(Stream *pStream)
{
    if (pStream == nullptr)
        return false;

    bool ok = pStream->read(&count, sizeof(count));
    if (ok && (count > 0))
    {
        triAccel = new TriangleFast[count];
        for (int i = 0; i < count && ok; i++)
        {
            ok |= triAccel[i].m_v0.deserialize(pStream);
            ok |= pStream->read(&triAccel[i].m_nu, sizeof(float));
            ok |= pStream->read(&triAccel[i].m_nv, sizeof(float));
            ok |= pStream->read(&triAccel[i].m_nd, sizeof(float));
            ok |= pStream->read(&triAccel[i].m_k, sizeof(int));
            ok |= pStream->read(&triAccel[i].m_bnu, sizeof(float));
            ok |= pStream->read(&triAccel[i].m_bnv, sizeof(float));
            ok |= pStream->read(&triAccel[i].m_cnu, sizeof(float));
            ok |= pStream->read(&triAccel[i].m_cnv, sizeof(float));
        }
    }

    return ok;
}

TriangleMesh::TriangleFast::TriangleFast(const Vector3f& v0, const Vector3f& v1, const Vector3f& v2)
{
    m_v0 = v0;

    // Find vectors for two edges sharing v0.
    Vector3f c(v1 - v0);
    Vector3f b(v2 - v0);

    // Compute normal
    Vector3f N = cross(b, c);

    // Identify primary plane
    if (fabs(N.x()) > fabs(N.y()))
    {
        if (fabs(N.x()) > fabs(N.z()))
            m_k = 0;
        else
            m_k = 2;
    }
    else
    {
        if (fabs(N.y()) > fabs(N.z()))
            m_k = 1;
        else
            m_k = 2;
    }

    // Compute triangle plane coefficients in projection plane
    int u = (m_k+1) % 3;
    int v = (m_k+2) % 3;
    float invNormPP = 1.0f / N[m_k];

    m_nu = N[u] * invNormPP;
    m_nv = N[v] * invNormPP;
    m_nd = dot(N, v0) * invNormPP;

    // Compute projection plane edge equations
    float invDet = 1.0f / (b[u] * c[v] - b[v] * c[u]);
    m_bnu = b[u] * invDet;
    m_bnv = -b[v] * invDet;

    m_cnu = c[v] * invDet;
    m_cnv = -c[u] * invDet;
}

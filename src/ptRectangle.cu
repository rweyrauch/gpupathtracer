#include "hip/hip_runtime.h"
/*
 * CUDA (GPU) Pathtracer based on Peter Shirley's 'Ray Tracing in One Weekend' e-book
 * series.
 *
 * Copyright (C) 2017 by Rick Weyrauch - rpweyrauch@gmail.com
 *
 * This code is licensed under the MIT license (MIT) (http://opensource.org/licenses/MIT)
 */

#include "ptRectangle.h"

bool XYRectangle::hit(const Rayf &r_in, float t0, float t1, HitRecord &rec, RNG &rng) const
{
    float t = (k - r_in.origin().z()) / r_in.direction().z();
    if (t < t0 || t > t1) return false;
    float x = r_in.origin().x() + t * r_in.direction().x();
    float y = r_in.origin().y() + t * r_in.direction().y();
    if (x < x0 || x > x1 || y < y0 || y > y1) return false;

    rec.uv.u() = (x - x0) / (x1 - x0);
    rec.uv.v() = (y - y0) / (y1 - y0);
    rec.t = t;
    rec.material = material;
    rec.p = r_in.pointAt(t);
    rec.normal = Vector3f(0, 0, 1);

    return true;
}

bool XYRectangle::serialize(Stream *pStream) const
{
    if (pStream == nullptr)
        return false;

    const int id = typeId();
    bool ok = pStream->write(&id, sizeof(id));
    if (material != nullptr)
        ok |= material->serialize(pStream);
    else
        ok |= pStream->writeNull();
    ok |= pStream->write(&x0, sizeof(x0));
    ok |= pStream->write(&x1, sizeof(x1));
    ok |= pStream->write(&y0, sizeof(y0));
    ok |= pStream->write(&y1, sizeof(y1));
    ok |= pStream->write(&k, sizeof(k));

    return ok;
}

bool XYRectangle::deserialize(Stream *pStream)
{
    if (pStream == nullptr)
        return false;

    material = Material::Create(pStream);
    bool ok = pStream->read(&x0, sizeof(x0));
    ok |= pStream->read(&x1, sizeof(x1));
    ok |= pStream->read(&y0, sizeof(y0));
    ok |= pStream->read(&y1, sizeof(y1));
    ok |= pStream->read(&k, sizeof(k));

    return ok;
}


bool XZRectangle::hit(const Rayf &r_in, float t0, float t1, HitRecord &rec, RNG &rng) const
{
    float t = (k - r_in.origin().y()) / r_in.direction().y();
    if (t < t0 || t > t1) return false;
    float x = r_in.origin().x() + t * r_in.direction().x();
    float z = r_in.origin().z() + t * r_in.direction().z();
    if (x < x0 || x > x1 || z < z0 || z > z1) return false;

    rec.uv.u() = (x - x0) / (x1 - x0);
    rec.uv.v() = (z - z0) / (z1 - z0);
    rec.t = t;
    rec.material = material;
    rec.p = r_in.pointAt(t);
    rec.normal = Vector3f(0, 1, 0);

    return true;
}

bool XZRectangle::serialize(Stream *pStream) const
{
    if (pStream == nullptr)
        return false;

    const int id = typeId();
    bool ok = pStream->write(&id, sizeof(id));
    if (material != nullptr)
        ok |= material->serialize(pStream);
    else
        ok |= pStream->writeNull();
    ok |= pStream->write(&x0, sizeof(x0));
    ok |= pStream->write(&x1, sizeof(x1));
    ok |= pStream->write(&z0, sizeof(z0));
    ok |= pStream->write(&z1, sizeof(z1));
    ok |= pStream->write(&k, sizeof(k));

    return ok;
}

bool XZRectangle::deserialize(Stream *pStream)
{
    if (pStream == nullptr)
        return false;

    material = Material::Create(pStream);
    bool ok = pStream->read(&x0, sizeof(x0));
    ok |= pStream->read(&x1, sizeof(x1));
    ok |= pStream->read(&z0, sizeof(z0));
    ok |= pStream->read(&z1, sizeof(z1));
    ok |= pStream->read(&k, sizeof(k));

    return ok;
}


bool YZRectangle::hit(const Rayf &r_in, float t0, float t1, HitRecord &rec, RNG &rng) const
{
    float t = (k - r_in.origin().x()) / r_in.direction().x();
    if (t < t0 || t > t1) return false;
    float y = r_in.origin().y() + t * r_in.direction().y();
    float z = r_in.origin().z() + t * r_in.direction().z();
    if (y < y0 || y > y1 || z < z0 || z > z1) return false;

    rec.uv.u() = (y - y0) / (y1 - y0);
    rec.uv.v() = (z - z0) / (z1 - z0);
    rec.t = t;
    rec.material = material;
    rec.p = r_in.pointAt(t);
    rec.normal = Vector3f(1, 0, 0);

    return true;
}

bool YZRectangle::serialize(Stream *pStream) const
{
    if (pStream == nullptr)
        return false;

    const int id = typeId();
    bool ok = pStream->write(&id, sizeof(id));
    if (material != nullptr)
        ok |= material->serialize(pStream);
    else
        ok |= pStream->writeNull();
    ok |= pStream->write(&y0, sizeof(y0));
    ok |= pStream->write(&y1, sizeof(y1));
    ok |= pStream->write(&z0, sizeof(z0));
    ok |= pStream->write(&z1, sizeof(z1));
    ok |= pStream->write(&k, sizeof(k));

    return ok;
}

bool YZRectangle::deserialize(Stream *pStream)
{
    if (pStream == nullptr)
        return false;

    material = Material::Create(pStream);
    bool ok = pStream->read(&y0, sizeof(y0));
    ok |= pStream->read(&y1, sizeof(y1));
    ok |= pStream->read(&z0, sizeof(z0));
    ok |= pStream->read(&z1, sizeof(z1));
    ok |= pStream->read(&k, sizeof(k));

    return ok;
}

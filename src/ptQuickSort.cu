#include "hip/hip_runtime.h"
/*
 * CUDA (GPU) Pathtracer based on Peter Shirley's 'Ray Tracing in One Weekend' e-book
 * series.
 *
 * Copyright (C) 2017 by Rick Weyrauch - rpweyrauch@gmail.com
 *
 * This code is licensed under the MIT license (MIT) (http://opensource.org/licenses/MIT)
 */

#include "ptQuickSort.h"

COMMON_FUNC void swap(Hitable** a, Hitable** b)
{
    Hitable* temp = *a;
    *a = *b;
    *b = temp;
}

COMMON_FUNC int partition(Hitable** list, int l, int h, int index)
{
    auto x = list[h];
    int i = (l - 1);

    for (int j = l; j <= h- 1; j++)
    {
        AABB<float> boxLeft, boxRight;
        list[j]->bounds(0, 0, boxLeft);
        x->bounds(0, 0, boxRight);

        if (boxLeft.min()[index] < boxRight.min()[index])
        {
            i++;
            swap(&list[i], &list[j]);
        }
    }
    swap(&list[i + 1], &list[h]);
    return (i + 1);
}

COMMON_FUNC void quickSort(Hitable** list, int l, int h, int index)
{
    // Create an auxiliary stack
    int* stack = new int[h - l + 1];

    // initialize top of stack
    int top = -1;

    // push initial values of l and h to stack
    stack[++top] = l;
    stack[++top] = h;

    // Keep popping from stack while is not empty
    while (top >= 0)
    {
        // Pop h and l
        h = stack[top--];
        l = stack[top--];

        // Set pivot element at its correct position
        // in sorted array
        int p = partition(list, l, h, index);

        // If there are elements on left side of pivot,
        // then push left side to stack
        if (p-1 > l)
        {
            stack[++top] = l;
            stack[++top] = p - 1;
        }

        // If there are elements on right side of pivot,
        // then push right side to stack
        if (p+1 < h)
        {
            stack[++top] = p + 1;
            stack[++top] = h;
        }
    }

    delete[] stack;
}